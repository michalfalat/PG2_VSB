#include "hip/hip_runtime.h"
#include "optixtutorial.h"


struct TriangleAttributes
{
	optix::float3 normal;
	optix::float2 texcoord;
};



rtBuffer<uchar4, 2> output_buffer;
rtBuffer<optix::float3> normal_buffer;
rtBuffer<optix::float2, 1> texcoord_buffer;
rtBuffer<optix::uchar1> material_index_buffer;


rtDeclareVariable( rtObject, top_object, , );
rtDeclareVariable( uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable( uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable( PerRayData_radiance, ray_data, rtPayload, );
rtDeclareVariable( float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(TriangleAttributes, attribs, attribute attributes, "Triangle attributes");
rtDeclareVariable(optix::float3, view_from, , );
rtDeclareVariable(optix::Matrix3x3, M_c_w, , "camera to world space transformation matrix" );
rtDeclareVariable(float, focal_length, , "focal length in pixels" );



RT_PROGRAM void primary_ray( void )
{
	if ( launch_index.x == 0 && launch_index.y % 100 == 0 )
	{
		//rtPrintf("(%u, %u)\n", launch_index.x, launch_index.y);
	}

	const optix::float3 d_c = make_float3(launch_index.x -
		launch_dim.x * 0.5f, output_buffer.size().y * 0.5f -
		launch_index.y, -focal_length);
	const optix::float3 d_w = optix::normalize(M_c_w * d_c);
	optix::Ray ray(view_from, d_w, 0, 0.01f);

	PerRayData_radiance prd;
	rtTrace(top_object, ray, prd);
	output_buffer[launch_index] = optix::make_uchar4(prd.result.x*255.0f, prd.result.y*255.0f, prd.result.z*255.0f, 255);

	/*const optix::float3 d_c = make_float3(launch_index.x - output_buffer.size().x * 0.5f, output_buffer.size().y * 0.5f - launch_index.y, -focal_length);
	const optix::float3 d_w = optix::normalize(M_c_w * d_c);
	optix::Ray ray(view_from, d_w, 0, 0.01f);

	//optix::Ray ray( optix::make_float3( launch_index.x, launch_index.y, 1.0f ),
		//optix::normalize( optix::make_float3( 0.0f, 0.0f, -1.0f ) ), 0, 0.01f );
	PerRayData_radiance prd;
	rtTrace( top_object, ray, prd );

	// access to buffers within OptiX programs uses a simple array syntax	
	output_buffer[launch_index] = optix::make_uchar4( prd.result.x*255.0f, prd.result.y*255.0f, prd.result.z*255.0f, 255.0f );*/
}

/*RT_PROGRAM void attribute_program(void)
{
	const optix::float2 barycentrics = rtGetTriangleBarycentrics();
	const unsigned int index = rtGetPrimitiveIndex();
	const optix::float3 n0 = normal_buffer[index * 3 + 0];
	const optix::float3 n1 = normal_buffer[index * 3 + 0];
	const optix::float3 n2 = normal_buffer[index * 3 + 0];
		attribs.normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y +
			n0 * (1.0f - barycentrics.x - barycentrics.y));
}*/


RT_PROGRAM void closest_hit( void )
{
	const unsigned int index = rtGetPrimitiveIndex();
	const optix::float3 n0 = normal_buffer[index * 3 + 0];
	const optix::float3 n1 = normal_buffer[index * 3 + 1];
	const optix::float3 n2 = normal_buffer[index * 3 + 2];
	optix::float3 normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y));

	ray_data.result = optix::make_float3((normal.x + 1) / 2, (normal.y + 1) / 2, (normal.z + 1) / 2);
}

/* may access variables declared with the rtPayload semantic in the same way as closest-hit and any-hit programs */
RT_PROGRAM void miss_program( void )
{
	ray_data.result = optix::make_float3( 0.0f, 0.0f, 1.0f );
}

RT_PROGRAM void exception( void )
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Exception 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y );
	rtPrintExceptionDetails();
	output_buffer[launch_index] = uchar4{ 255, 0, 255, 0 };
}
