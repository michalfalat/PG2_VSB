#include "hip/hip_runtime.h"
#include "optixtutorial.h"

rtBuffer<uchar4, 2> output_buffer;

rtDeclareVariable( rtObject, top_object, , );
rtDeclareVariable( uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable( uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable( PerRayData_radiance, ray_data, rtPayload, );
rtDeclareVariable( float2, barycentrics, attribute rtTriangleBarycentrics, );

RT_PROGRAM void primary_ray( void )
{
	if ( launch_index.x == 0 && launch_index.y % 100 == 0 )
	{
		rtPrintf( "(%u, %u)\n", launch_index.x, launch_index.y );	
	}	

	optix::Ray ray( optix::make_float3( launch_index.x, launch_index.y, 1.0f ),
		optix::normalize( optix::make_float3( 0.0f, 0.0f, -1.0f ) ), 0, 0.01f );
	PerRayData_radiance prd;
	rtTrace( top_object, ray, prd );

	// access to buffers within OptiX programs uses a simple array syntax	
	output_buffer[launch_index] = optix::make_uchar4( prd.result.x*255.0f, prd.result.y*255.0f, prd.result.z*255.0f, 255.0f );
}

RT_PROGRAM void closest_hit( void )
{
	ray_data.result = optix::make_float3( barycentrics.x, barycentrics.y, 0.0f );
}

/* may access variables declared with the rtPayload semantic in the same way as closest-hit and any-hit programs */
RT_PROGRAM void miss_program( void )
{
	ray_data.result = optix::make_float3( 0.0f, 0.0f, 1.0f );
}

RT_PROGRAM void exception( void )
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Exception 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y );
	rtPrintExceptionDetails();
	output_buffer[launch_index] = uchar4{ 255, 0, 255, 0 };
}
