#include "hip/hip_runtime.h"
#include "optixtutorial.h"

struct IntersectionInfo
{
	optix::float3 normal;
	optix::float2 texcoord;
	optix::float3 intersectionPoint;
	optix::float3 light;
};

enum class Shader : char { NORMAL = 1, LAMBERT = 2, PHONG = 3};

rtBuffer<optix::float3, 1> normal_buffer;
rtBuffer<optix::float2, 1> texcoord_buffer;
rtBuffer<optix::uchar4, 2> output_buffer;

rtDeclareVariable(optix::float3, diffuse, , "diffuse");
rtDeclareVariable(optix::float3, specular, , "specular");
rtDeclareVariable(optix::float3, ambient, , "ambient");
rtDeclareVariable(float, shininess, , "shininess");

rtDeclareVariable(int, tex_diffuse_id, , "diffuse texture id");

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(PerRayData_radiance, ray_data, rtPayload, );
rtDeclareVariable(PerRayData_shadow, shadow_ray_data, rtPayload, );
rtDeclareVariable(float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(IntersectionInfo, hitInfo, attribute attributes, "Intersection info");
rtDeclareVariable(optix::float3, view_from, , );
rtDeclareVariable(optix::Matrix3x3, M_c_w, , "camera to worldspace transformation matrix");
rtDeclareVariable(float, focal_length, , "focal length in pixels");


RT_PROGRAM void attribute_program(void)
{
	const optix::float3 lightPossition = optix::make_float3(50, 0, 120);
	const optix::float2 barycentrics = rtGetTriangleBarycentrics();
	const unsigned int index = rtGetPrimitiveIndex();
	const optix::float3 n0 = normal_buffer[index * 3 + 0];
	const optix::float3 n1 = normal_buffer[index * 3 + 1];
	const optix::float3 n2 = normal_buffer[index * 3 + 2];

	const optix::float2 t0 = texcoord_buffer[index * 3 + 0];
	const optix::float2 t1 = texcoord_buffer[index * 3 + 1];
	const optix::float2 t2 = texcoord_buffer[index * 3 + 2];

	hitInfo.normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y));
	hitInfo.texcoord = t1 * barycentrics.x + t2 * barycentrics.y + t0 * (1.0f - barycentrics.x - barycentrics.y);

	if (optix::dot(ray.direction, hitInfo.normal) > 0) {
		hitInfo.normal *= -1;
	}

	hitInfo.intersectionPoint = optix::make_float3(ray.origin.x + ray.tmax * ray.direction.x,
		ray.origin.y + ray.tmax * ray.direction.y,
		ray.origin.z + ray.tmax * ray.direction.z);

	hitInfo.light = optix::normalize(lightPossition - hitInfo.intersectionPoint);
}

RT_PROGRAM void primary_ray(void)
{
	PerRayData_radiance prd;
	hiprandState_t state;
	prd.state = &state;
	hiprand_init(launch_index.x + launch_dim.x * launch_index.y, 0, 0, prd.state);
	int ANTI_ALIASING_SAMPLES = 5;
	int NO_SAMPLES = 1;

	optix::float3 resultColor = optix::make_float3(0.0f, 0.0f, 0.0f);
	for (int i = 0; i < ANTI_ALIASING_SAMPLES; i++)
	{
		float randomX = hiprand_uniform(prd.state);
		float randomY = hiprand_uniform(prd.state);

		const optix::float3 d_c = make_float3(launch_index.x - launch_dim.x * 0.5f + randomX,
			output_buffer.size().y * 0.5f - launch_index.y + randomY,
			-focal_length);

		const optix::float3 d_w = optix::normalize(M_c_w * d_c);
		optix::Ray ray(view_from, d_w, 0, 0.01f);

		optix::float3 ambientColor = optix::make_float3(0.0f, 0.0f, 0.0f);
		for (int j = 0; j < NO_SAMPLES; j++) {
			rtTrace(top_object, ray, prd);
			ambientColor += prd.result;
		}
		ambientColor /= NO_SAMPLES;
		resultColor += ambientColor;
	}
	resultColor /= ANTI_ALIASING_SAMPLES;
	output_buffer[launch_index] = optix::make_uchar4(resultColor.x*255.0f, resultColor.y*255.0f, resultColor.z*255.0f, 255);
}

RT_PROGRAM void closest_hit_normal_shader(void)
{
	optix::float3 normal = hitInfo.normal;
	ray_data.result = optix::make_float3((normal.x + 1) / 2, (normal.y + 1) / 2, (normal.z + 1) / 2);
}


RT_PROGRAM void closest_hit_lambert_shader(void)
{
	float normalLigthScalarProduct = optix::dot(hitInfo.light, hitInfo.normal);
	ray_data.result = getDiffuseColor() * normalLigthScalarProduct * getAmbientColor();
}

RT_PROGRAM void closest_hit_phong_shader(void)
{
	float normalLigthScalarProduct = optix::dot(hitInfo.light, hitInfo.normal);

	optix::float3 lr = 2 * (normalLigthScalarProduct)* hitInfo.normal - hitInfo.light;
	ray_data.result.x = ambient.x + (getDiffuseColor().x * normalLigthScalarProduct) + specular.x * pow( optix::dot(-ray.direction, lr), shininess);
	ray_data.result.y = ambient.y + (getDiffuseColor().y * normalLigthScalarProduct) + specular.y * pow( optix::dot(-ray.direction, lr), shininess);
	ray_data.result.z = ambient.z + (getDiffuseColor().z * normalLigthScalarProduct) + specular.z * pow( optix::dot(-ray.direction, lr), shininess);

	ray_data.result = ray_data.result * getAmbientColor();

}

RT_PROGRAM void closest_hit_glass_shader(void)
{
}

RT_PROGRAM void closest_hit_pbr_shader(void)
{
}

RT_PROGRAM void closest_hit_mirror_shader(void)
{
}

RT_PROGRAM void any_hit(void)
{
	shadow_ray_data.visible.x = 0;
	rtTerminateRay();
}

RT_PROGRAM void miss_program(void)
{
	ray_data.result = optix::make_float3(0.0f, 0.0f, 0.0f);
}

RT_PROGRAM void exception(void)
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Exception 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y);
	rtPrintExceptionDetails();
	output_buffer[launch_index] = uchar4{ 255, 0, 255, 0 };
}


__device__ optix::float3 sampleHemisphere(optix::float3 normal, hiprandState_t* state, float& pdf) {
	float randomU = hiprand_uniform(state);
	float randomV = hiprand_uniform(state);

	float x = cosf(2 * HIP_PI_F * randomU) * sqrtf(1 - randomV);
	float y = sinf(2 * HIP_PI_F * randomU) * sqrtf(1 - randomV);
	float z = sqrtf(randomV);

	optix::float3 O1 = optix::normalize(orthogonal(normal));
	optix::float3 O2 = optix::normalize(optix::cross(normal, O1));

	optix::Matrix3x3 transformationMatrix = optix::make_matrix3x3(optix::Matrix<4, 4>::fromBasis(O1, O2, normal, optix::make_float3(0.0f, 0.0f, 0.0f)));

	optix::float3 omegai = optix::make_float3(x, y, z);

	omegai = optix::normalize(transformationMatrix * omegai);

	pdf = optix::dot(normal, omegai) / HIP_PI_F;

	return omegai;
}

__device__ optix::float3 orthogonal(const optix::float3 & v)
{
	return (abs(v.x) > abs(v.z)) ? optix::make_float3(-v.y, v.x, 0.0f) : optix::make_float3(0.0f, -v.z, v.y);
}

__device__ optix::float3 getAmbientColor()
{
	float pdf = 0;
	optix::float3 omegai = sampleHemisphere(hitInfo.normal, ray_data.state, pdf);

	optix::Ray ray(hitInfo.intersectionPoint, omegai, 1, 0.01f);
	PerRayData_shadow shadow_ray;
	shadow_ray.visible.x = 1;
	rtTrace(top_object, ray, shadow_ray);

	optix::float3 whiteColor = optix::make_float3(1, 1, 1);
	return whiteColor * optix::dot(hitInfo.normal, omegai) * shadow_ray.visible.x / HIP_PI_F / pdf;
}

__device__ optix::float3 getDiffuseColor()
{
	optix::float3 color;
	if (tex_diffuse_id != -1) {
		const optix::float4 value = optix::rtTex2D<optix::float4>(tex_diffuse_id, hitInfo.texcoord.x, 1 - hitInfo.texcoord.y);
		color = optix::make_float3(value.x, value.y, value.z);
	}
	else {
		color = diffuse;
	}

	return color;
}